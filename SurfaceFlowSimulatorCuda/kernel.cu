#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <device_atomic_functions.h>

#include "kernel.cuh"

#include <vector>
#include "SurfaceFlowSimulator.h"


template <typename T>
void copyToDevice(T* data, int size, T* gpuData)
{
	if (gpuData == nullptr) {
		hipMalloc(&gpuData, sizeof(T) * size);
	}
	hipMemcpy(gpuData, data, sizeof(T) * size, hipMemcpyHostToDevice);
}

void copyFlowTrack(std::vector<polyline3D>& flowTracks, point3D*& flowTrack, int*& lineSize, int& size)
{
	int sum = 0;
	int* cpuLineSize = new int[flowTracks.size()];
	std::vector<point3D> cpuLines;
	for (size_t i = 0; i < flowTracks.size(); ++i) {
		sum += flowTracks[i].size();
		cpuLineSize[i] = flowTracks[i].size();
		if (i > 0) {
			cpuLineSize[i] += cpuLineSize[i - 1];
		}
		for (size_t j = 0; j < flowTracks[i].size(); ++j) {
			cpuLines.emplace_back(flowTracks[i][j]);
		}
	}

	hipMalloc(&flowTrack, sizeof(point3D) * sum);
	hipMemcpy(flowTrack, cpuLines.data(), sizeof(point3D) * sum, hipMemcpyHostToDevice);
	hipMalloc(&lineSize, sizeof(int) * flowTracks.size());
	hipMemcpy(lineSize, cpuLineSize, sizeof(int) * flowTracks.size(), hipMemcpyHostToDevice);
	size = flowTracks.size();
}

void copyRainIdx(int endTime, int plineNum, int*& rainIdx)
{
	hipMalloc(&rainIdx, sizeof(int) * endTime * plineNum);
	hipMemset(rainIdx, 0, sizeof(int) * endTime * plineNum);
}

void copyFlowVal(int endTime, int plineNum, float*& flowVal)
{
	hipMalloc(&flowVal, sizeof(float) * endTime * plineNum);
	hipMemset(flowVal, 0, sizeof(float) * endTime * plineNum);
}

//void runKernel();

template <typename T>
void copyToHost(T* data, int size, T* hostData)
{
	if (hostData == nullptr) {
		hostData = new T[size];
	}
	hipMemcpy(hostData, data, sizeof(T) * size, hipMemcpyDeviceToHost);
}

template<typename T>
void copyDataHtD(T* cpuData, int size, T*& gpuData)
{
	if (cpuData == nullptr) {
		return;
	}
	hipMalloc(&gpuData, sizeof(T) * size);
	hipMemcpy(gpuData, cpuData, sizeof(T) * size);
}

__global__ void markRainIdx(point3D* flowTracks, int* flowTracksSize, int flowTracksSizeSize, double xmin, double ymax, double dx, double dy,
	float* pRVal, int imgDemWidth, int* rainIdx, float* flowVal, int plineNum, int endTime, int* rainIdxLen)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = threadIdx.y;
	int id = idy * (gridDim.x * blockDim.x) + idx;
	if (id >= plineNum)
		return;

	int plineStart = 0;
	if (id > 0)
		plineStart = flowTracksSize[id - 1];
	point3D runPnt = flowTracks[plineStart];
	
	int col = (runPnt.x - xmin) / dx;
	int row = (ymax - runPnt.y) / dy;

	if (col < 0) col = 0;
	if (row < 0) row = 0;

	float rValue = pRVal[row * imgDemWidth + col];
	if (rValue > 0) {
		int rainIdxEnd = id * endTime + rainIdxLen[id];
		rainIdx[rainIdxEnd] = 0;
		flowVal[rainIdxEnd] = rValue;
		rainIdxLen[id]++;
	}
}

void runMarkRainIdx(point3D* flowTracks, int* flowTracksSize, int flowTracksSizeSize, double xmin, double ymax, double dx, double dy,
	float* pRVal, int imgDemWidth, int* rainIdx, float* flowVal, int plineNum, int endTime, int* rainIdxLen)
{
	dim3 tpb(32, 32);
	int blockNum = (plineNum + 32 * 32 - 1) / (32 * 32);
	markRainIdx << <blockNum, tpb >> >(flowTracks, flowTracksSize, flowTracksSizeSize, xmin, ymax, dx, dy,
		pRVal, imgDemWidth, rainIdx, flowVal, plineNum, endTime, rainIdxLen);
}

__device__ void DeviceGetOneColor(colorTable* colors, int colorTableSize, float colDate, BYTE& colorR, BYTE& colorG, BYTE& colorB)
{
	for (int i = 0; i < colorTableSize; i++) {
		colorTable   aCol = colors[i];
		float starNum = aCol.starData;
		float endNum = aCol.endData;
		int colR = aCol.colorR;
		int colG = aCol.colorG;
		int colB = aCol.colorB;

		if (colDate >= starNum && colDate <= endNum) {
			colorR = (BYTE)colR;
			colorG = (BYTE)colG;
			colorB = (BYTE)colB;
			return;
		}
	}
}

__device__  inline void atomicFloatAdd(float *address, float val)
{
	int i_val = __float_as_int(val);
	int tmp0 = 0;
	int tmp1;

	while ((tmp1 = atomicCAS((int *)address, tmp0, i_val)) != tmp0) {
		tmp0 = tmp1;
		i_val = __float_as_int(val + __int_as_float(tmp1));
	}
}

//����һ����ε�Ԫ��runPnt��BMPλͼ�еı���pOutColor�����޸Ķ�Ӧդ��ļ���pOutNum��(�����ʵ�ʵ����������ۼ�)
__device__ void DeviceDrawOnePoint(point3D& runPnt, int imgWidth, int imgHeight, 
	double dx, double dy, double Xmin, double Ymax, float RVal,
	BYTE* pOutColor, colorTable* colors, int colorTableSize, float* pOutNum)
{
	if (runPnt.z <= 0)
		return;

	int col = (runPnt.x - Xmin) / dx;
	int row = (Ymax - runPnt.y) / dy;

	if (row <= 2 || row >= imgHeight - 2) return;
	if (col <= 2 || col >= imgWidth - 2) return;

	//pOutNum[row*imgWidth + col] = pOutNum[row*imgWidth + col] + RVal;
	atomicFloatAdd(&pOutNum[row*imgWidth + col], RVal);
	row = imgHeight - row;

	BYTE colorR = 0, colorG = 0, colorB = 255;
	DeviceGetOneColor(colors, colorTableSize, pOutNum[(imgHeight - row)*imgWidth + col], colorR, colorG, colorB);

	pOutColor[3 * ((row)*imgWidth + col) + 0] = colorG;   //R��ɫ  Ҳ�������GRB
	pOutColor[3 * ((row)*imgWidth + col) + 1] = colorR;   //G��ɫ
	pOutColor[3 * ((row)*imgWidth + col) + 2] = colorB;   //B��ɫ
}

__global__ void drawPoints(point3D* flowTracks, int* flowTracksSize, int flowTracksSizeSize, 
	int* rainIdx, float* flowVal, int plineNum, int endTime,
	int* rainIdxLen, int imgWidth, int imgHeight, double dx, double dy, double xmin, double ymax, 
	BYTE* outColor, colorTable* colorTables, int colorTableSize, float* pOutNum, int outNumSize)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = threadIdx.y;
	int id = idy * (gridDim.x * blockDim.x) + idx;
	if (id >= plineNum)
		return;

	int plineStart = 0;
	int plineSize = flowTracksSize[id];
	if (id > 0) {
		plineStart = flowTracksSize[id - 1];
		plineSize = plineSize - flowTracksSize[id - 1];
	}

	if (rainIdxLen[id] < 1 || plineSize < 1)
		return;

	for (int n = 0; n < rainIdxLen[id]; n++) {
		if (rainIdx[id * endTime + n] >= plineSize || rainIdx[id * endTime + n] < 0)
			continue;
		point3D runPnt = flowTracks[plineStart + rainIdx[id * endTime + n]];//flowTracks[rainIdx[id * endTime + n]];
		DeviceDrawOnePoint(runPnt, imgWidth, imgHeight, dx, dy, xmin, ymax, 
			flowVal[id * endTime + n], outColor, colorTables, colorTableSize, pOutNum);
		rainIdx[id * endTime + n] = rainIdx[id * endTime + n] + 1;
	}
}

void runDrawPoints(point3D* flowTracks, int* flowTracksSize, int flowTracksSizeSize, int* rainIdx, float* flowVal, int plineNum, int endTime,
	int* rainIdxLen, int imgWidth, int imgHeight, double dx, double dy, double xmin, double ymax, BYTE* outColor, colorTable* colorTables, int colorTableSize, float* pOutNum, int outNumSize)
{
	dim3 tpb(32, 32);
	int blockNum = (plineNum + 32 * 32 - 1) / (32 * 32);
	drawPoints << <blockNum, tpb >> >(flowTracks, flowTracksSize, flowTracksSizeSize, rainIdx, flowVal, plineNum, endTime,
		rainIdxLen, imgWidth, imgHeight, dx, dy, xmin, ymax, outColor, colorTables, colorTableSize, pOutNum, outNumSize);
}